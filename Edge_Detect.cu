#include "hip/hip_runtime.h"
2 # include <time .h>
3 # include <stdlib .h>
4 # include <stdio .h>
5 # include <string .h>
6 # include <math .h>
7 # include <cuda .h>
8 # include <cutil .h>
9 # include <ctime >
39
10
11 unsigned int width , height ;
12
13 int Gx [3][3] = { -1 , 0 , 1 ,
14 -2 , 0 , 2 ,
15 -1 , 0 , 1};
16
17 int Gy [3][3] = {1 ,2 ,1 ,
18 0 ,0 ,0 ,
19 -1 , -2 , -1};
20
21 int getPixel ( unsigned char * org , int col , int row) {
22
23 int sumX , sumY ;
24 sumX = sumY = 0;
25
26 for (int i= -1; i <= 1; i++) {
27 for (int j= -1; j <=1; j++) {
28 int curPixel = org [( row + j) * width + (col + i) ];
29 sumX += curPixel * Gx[i +1][ j +1];
30 sumY += curPixel * Gy[i +1][ j +1];
31 }
32 }
33 int sum = abs( sumY ) + abs( sumX ) ;
34 if (sum > 255) sum = 255;
35 if (sum < 0) sum = 0;
36 return sum ;
37 }
38
39 void h_EdgeDetect ( unsigned char * org , unsigned char * result ) {
40 int offset = 1 * width ;
41 for (int row =1; row < height -2; row ++) {
42 for (int col =1; col <width -2; col ++) {
43 result [ offset + col ] = getPixel (org , col , row ) ;
44 }
45 offset += width ;
46 }
47 }
48
40
49 __global__ void d_EdgeDetect ( unsigned char *org , unsigned char *
result , int width , int height ) {
50 int col = blockIdx .x * blockDim .x + threadIdx .x;
51 int row = blockIdx .y * blockDim .y + threadIdx .y;
52
53 if (row < 2 || col < 2 || row >= height -3 || col >= width -3 )
54 return ;
55
56 int Gx [3][3] = { -1 , 0 , 1 ,
57 -2 , 0 , 2 ,
58 -1 , 0 , 1};
59
60 int Gy [3][3] = {1 ,2 ,1 ,
61 0 ,0 ,0 ,
62 -1 , -2 , -1};
63
64 int sumX , sumY ;
65 sumX = sumY = 0;
66
67 for (int i= -1; i <= 1; i++) {
68 for (int j= -1; j <=1; j++) {
69 int curPixel = org [( row + j) * width + (col + i) ];
70 sumX += curPixel * Gx[i +1][ j +1];
71 sumY += curPixel * Gy[i +1][ j +1];
72 }
73 }
74
75 int sum = abs( sumY ) + abs( sumX ) ;
76 if (sum > 255) sum = 255;
77 if (sum < 0) sum = 0;
78
79 result [row * width + col ] = sum ;
80
81 }
82
83 int main ( int argc , char ** argv )
84 {
85 printf (" Starting program \n") ;
86
41
87 /* ******************** setup work ***************************
*/
88
89 unsigned char * d_resultPixels ;
90 unsigned char * h_resultPixels ;
91 unsigned char * h_pixels = NULL ;
92 unsigned char * d_pixels = NULL ;
93
94 char * srcPath = "/ Developer /GPU Computing /C/src / EdgeDetection /
image / cartoon .pgm";
95 char * h_ResultPath = "/ Developer /GPU Computing /C/src /
EdgeDetection / output / h_cartoon .pgm";
96 char * d_ResultPath = "/ Developer /GPU Computing /C/src /
EdgeDetection / output / d_cartoon .pgm";
97
98 cutLoadPGMub ( srcPath , & h_pixels , &width , & height ) ;
99
100 int ImageSize = sizeof ( unsigned char ) * width * height ;
101
102 h_resultPixels = ( unsigned char *) malloc ( ImageSize ) ;
103 hipMalloc (( void **) & d_pixels , ImageSize ) ;
104 hipMalloc (( void **) & d_resultPixels , ImageSize ) ;
105 hipMemcpy ( d_pixels , h_pixels , ImageSize , hipMemcpyHostToDevice
) ;
106
107 /* ******************** END setup work
*************************** */
108
109 /* ************************ Host processing
************************* */
110 clock_t starttime , endtime , difference ;
111
112 printf (" Starting host processing \n") ;
113 starttime = clock () ;
114 h_EdgeDetect ( h_pixels , h_resultPixels ) ;
115 endtime = clock () ;
116 printf (" Completed host processing \n") ;
117
118 difference = ( endtime - starttime ) ;
42
119 double interval = difference / ( double ) CLOCKS_PER_SEC ;
120 printf ("CPU execution time = %f ms\n", interval * 1000) ;
121 cutSavePGMub ( h_ResultPath , h_resultPixels , width , height ) ;
122 /* ************************ END Host processing
************************* */
123
124 /* ************************ Device processing
************************* */
125 dim3 block (16 ,16) ;
126 dim3 grid ( width /16 , height /16) ;
127 unsigned int timer = 0;
128 cutCreateTimer (& timer ) ;
129
130 printf (" Invoking Kernel \n") ;
131 cutStartTimer ( timer ) ;
132 /* CUDA method */
133 d_EdgeDetect <<< grid , block > > >( d_pixels , d_resultPixels , width
, height ) ;
134 hipDeviceSynchronize () ;
135 cutStopTimer ( timer ) ;
136 printf (" Completed Kernel \n") ;
137
138 printf (" CUDA execution time = %f ms\n", cutGetTimerValue ( timer ) )
;
139
140 hipMemcpy ( h_resultPixels , d_resultPixels , ImageSize ,
hipMemcpyDeviceToHost ) ;
141 cutSavePGMub ( d_ResultPath , h_resultPixels , width , height ) ;
142
143 /* ************************ END Device processing
************************* */
144
145
146
147 printf (" Press enter to exit ...\ n") ;
148 getchar () ;
149 }
